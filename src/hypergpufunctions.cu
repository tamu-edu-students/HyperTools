#include "hip/hip_runtime.h"
#pragma once
#include <opencv2/opencv.hpp>
#include <vector>
#include <thread>
#include "hyperfunctions.cpp"
#include "hyperfunctions.h"
#include "hypergpufunctions.h"
#include "ctpl.h"

using namespace cv;
using namespace std;
using namespace std::chrono;

/**
 * Measures spectral similarity between our image and a reference spectrum
 * with the Spectral Angle Mapper algorithm using concurrent GPU threads.
 * 
 * Formula : 
 * r^2 = sum of squared reference spectra values, 
 * i * r = sum of image spectra values * corresponding reference spectra values
 * i^2 = sum of squared image spectra values. 
 * 
 * returns : inverse cos of i * r / (r^2 * i^2). 
 * 
*/
__global__ void img_test_multi_thread_SAM(int *out, int *img_array, int n, int num_layers, int* ref_spectrum) 
{
    
    // parallelize tasks
    // pixels are stored with all pixel values next to each other for the layers    
    // n is number of pixels 
    // blockID : block index within the grid
    // blockDim : how many threads per block
    // threadIdx : thread index within the block 

    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    float sum1=0, sum2=0;
    float sum3 = 0;
    for (int a=0; a<num_layers-1; a++) {
        sum3+=ref_spectrum[a] *ref_spectrum[a]; //sum of squared reference spectra values
    }
    if (tid < n){
        int offset=tid*num_layers; //calculating which index in the image array the values for threadID pixel start at
        for (int a=0; a<num_layers-1; a++) //iterating through spectra layers for that pixel
        {
            sum1+=img_array[offset+a]*ref_spectrum[a]; //image spectra values * corresponding referencec spectrum values
            sum2+=img_array[offset+a]*img_array[offset+a]; //Squared image spectra values
        }
        
        if (sum1<=0 || sum2<=0 || sum3<=0 )
        {
            out[tid] =255; // set to white due to an error
        }
        else
        {
            float temp1= sum1/(sqrt(sum2)*sqrt(sum3));
            double alpha_rad=acos(temp1);
            out[tid] =(int)((double)alpha_rad*(double)255/(double)3.14159) ;
        }
    }
    
}

/**
 * Measures spectral similarity between our image and a reference spectrum
 * with the Spectral Information Divergence algorithm using concurrent GPU threads.
 * 
 * Formula : 
 * q = probability array for our image, divide image array by sum of image array values.
 * p = probability array for reference spectrum, divide reference spectrum array by sum of reference spectrum array values
 * 
 * returns : sum of p[i] * log(p[i]/q[i]) + sum of q[i] * log(q[i]/p[i]) for 0 <= i < num_layers-1 
 * 
*/

__global__ void img_test_multi_thread_SID(int *out, int *img_array, int n, int num_layers, int* ref_spectrum) 
{
    // parallelize tasks
    
    // pixels are stored with all pixel values next to eachother for the layers    
    // n is number of pixels 
    // blockID : block index within the grid
    // blockDim : how many threads per block
    // threadIdx : thread index within the block 

    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    float sum1=0, sum2=0, ref_sum=0, pix_sum=0;

    if (tid < n){
        int offset=tid*num_layers;
        for (int a=0; a<num_layers-1; a++)
        {
            if (ref_spectrum[a]<1){ref_spectrum[a]+=1;}
            if (img_array[offset+a]<1){img_array[offset+a]+=1;}              
            ref_sum+= ref_spectrum[a] ;
            pix_sum+= img_array[offset+a];
        }
        
        // error handling to avoid division by zero
        if (ref_sum<1){ref_sum+=1;}
        if (pix_sum<1){pix_sum+=1;}
        
        float ref_new[300], pix_new[300];
        
        for (int a=0; a<num_layers-1; a++)
        {
            ref_new[a]=ref_spectrum[a] / ref_sum ; //probability distribution for reference spectrum
            pix_new[a]=img_array[offset+a]/pix_sum; //probabiltiy distribution for our image
            // error handling to avoid division by zero
        }
        
        for (int a=0; a<num_layers-1; a++)
        {
            sum1+= ref_new[a]*log(ref_new[a]/pix_new[a]);
            sum2+= pix_new[a]*log(pix_new[a]/ref_new[a]);
        }        

        // need to normalize the results better here
        out[tid] =(sum1+sum2) *60;
        if (out[tid]>255){out[tid]=255;}

    }
    
}

/**
 * 
 * Spectral Corellation Mapper function for spectral similarity analysis
 * 
 * 
 * 
*/
__global__ void img_test_multi_thread_SCM(int *out, int *img_array, int n, int num_layers, int* ref_spectrum) 
{ 
    // parallelize tasks
    // pixels are stored with all pixel values next to eachother for the layers    
    // n is number of pixels 

    // blockID : block index within the grid
    // blockDim : how many threads per block
    // threadIdx : thread index within the block 

    int tid = blockIdx.x * blockDim.x + threadIdx.x; //unique thread ID
    float sum1=0, sum2=0, sum3=0, mean1=0, mean2=0;
    if (tid < n){
        int offset=tid*num_layers;

        for (int a=0; a<num_layers-1; a++)
        {
            mean1+=((float)1/(float)(num_layers-1)* (float)img_array[offset+a])  ;
            mean2+=((float)1/(float)(num_layers-1)* (float)ref_spectrum[a]) ;
        }

        for (int a=0; a<num_layers-1; a++)
        {
            sum1+=(img_array[offset+a]-mean1)*(ref_spectrum[a]-mean2) ;
            sum2+=(img_array[offset+a]-mean1)*(img_array[offset+a]-mean1);
            sum3+=(ref_spectrum[a]-mean2)*(ref_spectrum[a]-mean2);
        }        
        if (sum2<=0 || sum3<=0 )
        {
            out[tid] =255; // set to white due to an error
        }
        else
        {
            float temp1= sum1/(sqrt(sum2)*sqrt(sum3));
            double alpha_rad=acos(temp1);
            out[tid] =(int)((double)alpha_rad*(double)255/(double)3.14159) ;
        }
    }
}

/**
 * Calls the multithreaded spectral similarity algorithms, based on the variable spec_sim_alg, set in
    hyperfunctions.cpp.
 * Retrieves output in "out", then calls oneD_array_to_mat(out) to convert out into the OPENCV matrix "spec_simil_img".
*/

void HyperFunctionsGPU::spec_sim_GPU() {

    if (spec_sim_alg == 0) { //running the multithreaded algorithms
        img_test_multi_thread_SAM<<<grid_size,block_size>>>(d_out, d_img_array, N_size, num_lay, d_ref_spectrum);
    } else if (spec_sim_alg == 1) {
        img_test_multi_thread_SCM<<<grid_size,block_size>>>(d_out, d_img_array, N_size, num_lay, d_ref_spectrum);
    } else if (spec_sim_alg == 2) {
        img_test_multi_thread_SID<<<grid_size,block_size>>>(d_out, d_img_array, N_size, num_lay, d_ref_spectrum);
    }

    hipDeviceSynchronize();
    hipMemcpyAsync(out, d_out, sizeof(int) * N_size, hipMemcpyDeviceToHost); 
    hipDeviceSynchronize();

    this->oneD_array_to_mat(out);   
}

void HyperFunctionsGPU::deallocate_memory() 
{
    hipFree(d_ref_spectrum); hipFree(d_out); hipHostFree(out);
}

/* allocating CUDA memory. 
* In cuda, grids contain blocks of threads, which are used for parallel computations. 
* One pixel in img_array needs one thread for a computation, so threads = number of pixels. 
* Set the number of threads per block to be 512 - this is the maximum threads per block for older GPUs.
* Grid size is the number of of blocks, given by the number of threads / block size + 1. 
*/
void HyperFunctionsGPU::allocate_memory() {
    N_points=mlt1[1].rows*mlt1[1].cols*mlt1.size(); 
    N_size=mlt1[1].rows*mlt1[1].cols;    
    num_lay=  mlt1.size();
    block_size = 512;
    grid_size = ((N_points + block_size) / block_size); 
    
    int tmp_len1=reference_spectrums[ref_spec_index].size(); 

    hipHostAlloc ((void**)&out, sizeof(int) * N_size, hipHostMallocDefault);
    hipMalloc((void**)&d_out, sizeof(int) * N_size);
    hipMalloc((void**)&d_ref_spectrum, sizeof(int) * tmp_len1);

    //allocating memory on the GPU device

    ref_spectrum=new int[tmp_len1];
    for (int i=0;i<reference_spectrums[ref_spec_index].size();i++)
    {
        ref_spectrum[i] = reference_spectrums[ref_spec_index][i]; 
        //converting the 2-D array of reference spectrums into one-D for CUDA processing
    }
    hipMemcpy(d_ref_spectrum, ref_spectrum, sizeof(int) * tmp_len1, hipMemcpyHostToDevice);
    delete[] ref_spectrum;
    //copying our existing memory that holds image data and reference spectrum data to the 
    //allocated memory on the GPU
}

/**
 * Converts one-D array to 1 channel 8 bit OPENCV matrix. 
 * Used in manipulating spectral similarity data. 
*/
void HyperFunctionsGPU::oneD_array_to_mat(int* img_array)
{
    spec_simil_img = cv::Mat(mlt1[1].rows, mlt1[1].cols, CV_32SC1, img_array);
    spec_simil_img.convertTo(spec_simil_img, CV_8UC1); //converting to 8 bit unsigned, 1 channel. 
}

/**
 * Converting a one-D array to a OPENCV matrix. 
 * This matrix will has three channels per point which store RGB color values. 
*/

void HyperFunctionsGPU::oneD_array_to_mat(int* img_array, int cols, int rows, int channels, Mat* mlt1)
{

    *mlt1 = cv::Mat(mlt1->rows, mlt1->cols, CV_32SC3, img_array);
    mlt1->convertTo(*mlt1, CV_8UC3);
}


__global__ void mat_to_oneD_array_child(uchar* mat_array, int* img_array, int n, int start, int inc) 
{ 

    // blockID : block index within the grid
    // blockDim : how many threads per block
    // threadIdx : thread index within the block 
    int tid = blockIdx.x * blockDim.x + threadIdx.x; //unique thread ID
    if (tid < n){
       img_array[tid * inc + start] = mat_array[tid];
    }
}

void HyperFunctionsGPU::mat_to_oneD_array_parallel_parent()
{
    int array_size=mlt1[1].rows*mlt1[1].cols*mlt1.size();    
    uchar* d_mat_array; 
    int sz = mlt1[1].rows * mlt1[1].cols;
    hipHostAlloc ((void**)&d_mat_array, sizeof(uchar) * sz, hipHostMallocDefault);
    hipMalloc((void**)&d_img_array, sizeof(int) * array_size);
    int grid_size1 = (sz + block_size) / block_size;
    for (int i = 0; i < mlt1.size(); i++) {
        uchar* mat_array = (uchar*)mlt1[i].data;
        hipMemcpyAsync(d_mat_array, mat_array, sizeof(uchar) * sz, hipMemcpyHostToDevice);
        mat_to_oneD_array_child<<<grid_size1, block_size>>>(d_mat_array, d_img_array, sz, i, mlt1.size());
    }
    hipHostFree(d_mat_array);  
}

void HyperFunctionsGPU::mat_to_oneD_array_parallel_parent(vector<Mat>* matvector1)
{
    vector<Mat> matvector = *matvector1;
    uchar* d_mat_array;
    int array_size=matvector[1].rows*matvector[1].cols*matvector.size();  
    int sz = matvector[1].rows * matvector[1].cols;
    hipHostAlloc ((void**)&d_mat_array, sizeof(uchar) * sz, hipHostMallocDefault);
    hipMalloc((void**)&d_classified_img_array, sizeof(int) * array_size);
    block_size = 512;
    int grid_size1 = (sz + block_size) / block_size;
    for (int i = 0; i < matvector.size(); i++) {
        uchar* mat_array = (uchar*)matvector[i].data;
        hipMemcpy(d_mat_array, mat_array, sizeof(uchar) * sz, hipMemcpyHostToDevice);
        mat_to_oneD_array_child<<<grid_size1, block_size>>>(d_mat_array, d_classified_img_array, sz, i, matvector.size());
    }
    hipHostFree(d_mat_array);  
}

/**
 * Fills in the "out" array, which holds RGB values based on which reference spectra a pixel in the image is most similar to. 
*/

__global__ void img_test_classifier(int *out, int *img_array, int num_pixels, int num_spectrums, int* color_info, int classification_threshold ) 
{
    // blockID : block index within the grid
    // blockDim : how many threads per block
    // threadIdx : thread index within the block 

    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < num_pixels){
        //out[tid] =img_array[tid];
        int offset=tid*num_spectrums; //where the spectral similarity image starts for a pixel. 
        int low_val;
        for (int a=0; a<num_spectrums;a++) //iterating through all the spectral similarity scores for a pixel
        {
            if (a==0) 
            //Setting the initial lowest value. Goal is to look through all the spectral
            //similarity values for a pixel and find the lowest score. The reference
            //spectra that yields the lowest score will be the most similar.  
            {
                low_val=img_array[offset];
                if (low_val<=classification_threshold)
                {
                out[tid*3]=color_info[a+0];
                out[tid*3+1]=color_info[a+1];  
                out[tid*3+2]=color_info[a+2];
                }
                else
                {
                out[tid*3]=0;
                out[tid*3+1]=0;  
                out[tid*3+2]=0;
                }
            }
            else if (img_array[offset+a]<low_val && img_array[offset+a]<= classification_threshold) 
            //looking for a new minimum. If found, set the color channels of the RGB image to the color corresponding to the reference spectra. 
            {
                out[tid*3]=color_info[a*3+0];
                out[tid*3+1]=color_info[a*3+1];
                out[tid*3+2]=color_info[a*3+2];                
            }            
           
        }
    }   
}

/**
 * Takes hyperspectral data and gives each pixel a color based on which reference spectra
 * it is most similar to. 
 * 
*/

void HyperFunctionsGPU::semantic_segmentation() {
    ref_spec_index = 0;
    int tmp_len1 = reference_spectrums[0].size();
    vector<Mat> similarity_images;
    int* ref_spectrum = new int[tmp_len1];

    for (int i=0;i<reference_spectrums[ref_spec_index].size();i++)
    {
        ref_spectrum[i] = reference_spectrums[ref_spec_index][i]; 
    }
    hipMemcpy(d_ref_spectrum, ref_spectrum, sizeof(int) * tmp_len1, hipMemcpyHostToDevice);

    this->spec_sim_GPU(); 
    //performs spectral similarity, comparing each pixel in our image array to the first reference spectra. 
    similarity_images.push_back(spec_simil_img);  
    for (int i = 1; i < reference_spectrums.size(); i++) { //loop to iterate through all the reference spectras. 
        for (int j=0; j < reference_spectrums[i].size(); j++) {
            ref_spectrum[j] = reference_spectrums[i][j]; //updating the reference spectrum we are comparing our pixels to. 
        }
        //updating the memory allocated on the GPU that stores the current reference spectra
        hipMemcpy(d_ref_spectrum, ref_spectrum, sizeof(int) * tmp_len1, hipMemcpyHostToDevice); 
        this->spec_sim_GPU();
        similarity_images.push_back(spec_simil_img);
    }

    int array_size2=similarity_images[1].rows*similarity_images[1].cols*similarity_images.size();  
    //converting the vector of matrices that store the similarity values for each reference spectrum into a 1-D array

    mat_to_oneD_array_parallel_parent(&similarity_images);

    int *d_color_info, *d_out2, *out2;
    int N_size_sim = similarity_images[1].rows*similarity_images[1].cols*3; 
    int N_points_sim = similarity_images[1].rows*similarity_images[1].cols*similarity_images.size(); 
    int grid_size_sim = ((N_points + block_size) / block_size);
    int tmp_len2 = color_combos.size()*3;


    /**
     * out2 : 1-d array that represents matrix with 3 channels, to store R G and B values. 
     * d_clasified_img_array : 1-d array containing spectra similarity values 
     * d_color_info : holds reference colors that will be used to colorize our final image
     * 
    */
   
    hipHostAlloc ((void**)&out2, sizeof(int) *N_size_sim, hipHostMallocDefault); 
    hipMalloc((void**)&d_out2, sizeof(int) * N_size_sim);
    int temp_val=reference_colors.size() * 3;
    hipMalloc((void**)&d_color_info, sizeof(int) * temp_val);

    int* reference_colors_c = new int[reference_colors.size() * 3];

    //converting reference_colors into a 1-d array
    for (int i = 0; i < reference_colors.size(); i++) {
        reference_colors_c[i*3] = reference_colors[i][0];
        reference_colors_c[i*3+1] = reference_colors[i][1];
        reference_colors_c[i*3+2] = reference_colors[i][2];
    }

    hipMemcpy(d_color_info, reference_colors_c, sizeof(int) * temp_val, hipMemcpyHostToDevice);
    //multi-threaded function to find the most similar spectra for a pixel and color it based on the color assigned to that spectra
    img_test_classifier<<<grid_size_sim,block_size>>>(d_out2, d_classified_img_array, N_size_sim/3, similarity_images.size(), d_color_info,classification_threshold);

    /**
     * copying the color image into out2, and converting that into a OPENCV matrix. 
    */

    hipMemcpy(out2, d_out2, sizeof(int) * N_size_sim, hipMemcpyDeviceToHost);
   
    Mat test_img2(similarity_images[1].rows, similarity_images[1].cols, CV_8UC3, Scalar(0,0,0)); 
    oneD_array_to_mat(out2, similarity_images[1].cols,similarity_images[1].rows,3, &test_img2);
    classified_img = test_img2;

    hipFree(d_color_info);
    hipFree(d_out2);
    hipFree(d_classified_img_array);
    hipHostFree(out2);
    delete[] reference_colors_c;
    delete[] ref_spectrum;

}


