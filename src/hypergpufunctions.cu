#include "hip/hip_runtime.h"
#pragma once
#include <opencv2/opencv.hpp>
#include <vector>
#include <thread>
#include "hyperfunctions.cpp"
#include "hyperfunctions.h"
#include "hypergpufunctions.h"
#include "ctpl.h"

using namespace cv;
using namespace std;
using namespace std::chrono;

/**
 * Measures spectral similarity between our image and a reference spectrum
 * with the Spectral Angle Mapper algorithm using concurrent GPU threads.
 * 
 * Formula : 
 * r^2 = sum of squared reference spectra values, 
 * i * r = sum of image spectra values * corresponding reference spectra values
 * i^2 = sum of squared image spectra values. 
 * 
 * returns : inverse cos of i * r / (r^2 * i^2). 
 * 
*/
__global__ void img_test_multi_thread_SAM(float *out, int *img_array, int n, int num_layers, int* ref_spectrum) 
{
    
    // parallelize tasks
    // pixels are stored with all pixel values next to each other for the layers    
    // n is number of pixels 
    // blockID : block index within the grid
    // blockDim : how many threads per block
    // threadIdx : thread index within the block 

    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    float sum1=0, sum2=0;
    float sum3 = 0;
    for (int a=0; a<num_layers-1; a++) {
        sum3+=ref_spectrum[a] *ref_spectrum[a]; //sum of squared reference spectra values
    }
    if (tid < n){
        int offset=tid*num_layers; //calculating which index in the image array the values for threadID pixel start at
        for (int a=0; a<num_layers-1; a++) //iterating through spectra layers for that pixel
        {
            sum1+=img_array[offset+a]*ref_spectrum[a]; //image spectra values * corresponding referencec spectrum values
            sum2+=img_array[offset+a]*img_array[offset+a]; //Squared image spectra values
        }
        
        if (sum1<=0 || sum2<=0 || sum3<=0 )
        {
            out[tid] =255; // set to white due to an error
        }
        else
        {
            float temp1= sum1/(sqrt(sum2)*sqrt(sum3));
            double alpha_rad=acos(temp1);
            out[tid] =(int)((double)alpha_rad*(double)255/(double)3.14159) ;
        }
    }
    
}

/**
 * Measures spectral similarity between our image and a reference spectrum
 * with the Spectral Information Divergence algorithm using concurrent GPU threads.
 * 
 * Formula : 
 * q = probability array for our image, divide image array by sum of image array values.
 * p = probability array for reference spectrum, divide reference spectrum array by sum of reference spectrum array values
 * 
 * returns : sum of p[i] * log(p[i]/q[i]) + sum of q[i] * log(q[i]/p[i]) for 0 <= i < num_layers-1 
 * 
*/

__global__ void img_test_multi_thread_SID(float *out, int *img_array, int n, int num_layers, int* ref_spectrum) 
{
    // parallelize tasks
    
    // pixels are stored with all pixel values next to eachother for the layers    
    // n is number of pixels 
    // blockID : block index within the grid
    // blockDim : how many threads per block
    // threadIdx : thread index within the block 

    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    float sum1=0, sum2=0, ref_sum=0, pix_sum=0;

    if (tid < n){
        int offset=tid*num_layers;
        for (int a=0; a<num_layers-1; a++)
        {
            if (ref_spectrum[a]<1){ref_spectrum[a]+=1;}
            if (img_array[offset+a]<1){img_array[offset+a]+=1;}              
            ref_sum+= ref_spectrum[a] ;
            pix_sum+= img_array[offset+a];
        }
        
        // error handling to avoid division by zero
        if (ref_sum<1){ref_sum+=1;}
        if (pix_sum<1){pix_sum+=1;}
        
        float ref_new[300], pix_new[300];
        
        for (int a=0; a<num_layers-1; a++)
        {
            ref_new[a]=ref_spectrum[a] / ref_sum ; //probability distribution for reference spectrum
            pix_new[a]=img_array[offset+a]/pix_sum; //probabiltiy distribution for our image
            // error handling to avoid division by zero
        }
        
        for (int a=0; a<num_layers-1; a++)
        {
            sum1+= ref_new[a]*log(ref_new[a]/pix_new[a]);
            sum2+= pix_new[a]*log(pix_new[a]/ref_new[a]);
        }        

        // need to normalize the results better here
        out[tid] =(sum1+sum2) *60;
        if (out[tid]>255){out[tid]=255;}

    }
    
}

/**
 * 
 * Spectral Corellation Mapper function for spectral similarity analysis
 * 
 * 
 * 
*/
__global__ void img_test_multi_thread_SCM(float *out, int *img_array, int n, int num_layers, int* ref_spectrum) 
{ 
    // parallelize tasks
    // pixels are stored with all pixel values next to eachother for the layers    
    // n is number of pixels 

    // blockID : block index within the grid
    // blockDim : how many threads per block
    // threadIdx : thread index within the block 

    int tid = blockIdx.x * blockDim.x + threadIdx.x; //unique thread ID
    float sum1=0, sum2=0, sum3=0, mean1=0, mean2=0;
    if (tid < n){
        int offset=tid*num_layers;

        for (int a=0; a<num_layers-1; a++)
        {
            mean1+=((float)1/(float)(num_layers-1)* (float)img_array[offset+a])  ;
            mean2+=((float)1/(float)(num_layers-1)* (float)ref_spectrum[a]) ;
        }

        for (int a=0; a<num_layers-1; a++)
        {
            sum1+=(img_array[offset+a]-mean1)*(ref_spectrum[a]-mean2) ;
            sum2+=(img_array[offset+a]-mean1)*(img_array[offset+a]-mean1);
            sum3+=(ref_spectrum[a]-mean2)*(ref_spectrum[a]-mean2);
        }        
        if (sum2<=0 || sum3<=0 )
        {
            out[tid] =255; // set to white due to an error
        }
        else
        {
            float temp1= sum1/(sqrt(sum2)*sqrt(sum3));
            double alpha_rad=acos(temp1);
            out[tid] =(int)((double)alpha_rad*(double)255/(double)3.14159) ;
        }
    }
}

/**
 * Cosine Similiarity Algorithm
*/
__global__ void img_test_multi_thread_cos(float *out, int *img_array, int n, int num_layers, int* ref_spectrum) 
{
    
    // parallelize tasks
    // pixels are stored with all pixel values next to each other for the layers    
    // n is number of pixels 
    // blockID : block index within the grid
    // blockDim : how many threads per block
    // threadIdx : thread index within the block 

    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    float sum1=0, sum2=0;
    float sum3 = 0;
    for (int a=0; a<num_layers-1; a++) {
        sum3+=ref_spectrum[a] *ref_spectrum[a]; //sum of squared reference spectra values
    }
    if (tid < n){
        int offset=tid*num_layers; //calculating which index in the image array the values for threadID pixel start at
        for (int a=0; a<num_layers-1; a++) //iterating through spectra layers for that pixel
        {
            sum1+=img_array[offset+a]*ref_spectrum[a]; //image spectra values * corresponding referencec spectrum values
            sum2+=img_array[offset+a]*img_array[offset+a]; //Squared image spectra values
        }
        
        if (sum1<=0 || sum2<=0 || sum3<=0 )
        {
            out[tid] =255; // set to white due to an error
        }
        else
        {
            float temp1= sum1/(sqrt(sum2)*sqrt(sum3));
            double alpha_rad=acos(temp1);
            out[tid] =(int)((double)alpha_rad*(double)255) ;
        }
    }
    
}

/**
 * JM distance algorithm 
 * 
*/
__global__ void img_test_multi_thread_JM(int* out, int* img_array, int n, int num_layers, int* ref_spectrum) {

    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < n) {
        double referenceSpecIntegral = 0;
        double pixelSpecIntegral = 0;

        int offset = tid * num_layers;
        for (int i = 0; i < num_layers; i++) {
            referenceSpecIntegral += ref_spectrum[i];
            pixelSpecIntegral += img_array[offset + i];
        }

        double BC = 0;
        for (int i = 0; i < num_layers; i++) {
            BC += sqrt((ref_spectrum[i] / referenceSpecIntegral) * (img_array[offset + i] / pixelSpecIntegral));
        }

        double Bhattacharyya = -log(BC);
        double JM_distance = sqrt(2 * (1 - exp(-Bhattacharyya)));
        double JM_distance_scaled = JM_distance * 180.312229203;
        out[tid] = (int)(JM_distance_scaled);
    }
}

/**
 * City block algorithm 
 * 
*/
__global__ void img_test_multi_thread_cityblock(int* out, int* img_array, int n, int num_layers, int* ref_spectrum) {

    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    float sum1=0;
    
    if (tid < n){
        int offset = tid * num_layers;
        for (int a=0; a<num_layers-1; a++) {
            sum1 += abs(img_array[offset + a] - ref_spectrum[a]);

        }
        out[tid] = sum1/(num_layers + 255);
        
        
    }

}

/**
 * Euclidian
*/
__global__ void img_test_multi_thread_EuD(float *out, int *img_array, int n, int num_layers, int* ref_spectrum){
    
    // parallelize tasks
    // pixels are stored with all pixel values next to each other for the layers    
    // n is number of pixels 
    // blockID : block index within the grid
    // blockDim : how many threads per block
    // threadIdx : thread index within the block 

    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    float sum1=0, sum2=0, sum3 = 0;

    for (int a=0; a<num_layers-1; a++) {
        sum3+=ref_spectrum[a] *ref_spectrum[a]; //sum of squared reference spectra values
    }
    if (tid < n){
        int offset=tid*num_layers; //calculating which index in the image array the values for threadID pixel start at
        for (int a=0; a<num_layers-1; a++) //iterating through spectra layers for that pixel
        {
            sum1+=img_array[offset+a]*ref_spectrum[a]; //image spectra values * corresponding referencec spectrum values
            sum2+=img_array[offset+a]*img_array[offset+a]; //Squared image spectra values
        }
        
        if (sum1<=0 || sum2<=0 || sum3<=0 )
        {
            out[tid] =255; // set to white due to an error
        }
        else
        {
            float temp1= sum1/(sqrt(sum2)*sqrt(sum3));
            double alpha_rad=acos(temp1);
            temp1 = sin(alpha_rad/2);
            out[tid] =(int)((double)alpha_rad*(double)255) ;
        }
}
}
/**
 * Calls the multithreaded spectral similarity algorithms, based on the variable spec_sim_alg, set in
    hyperfunctions.cpp.
 * Retrieves output in "out", then calls oneD_array_to_mat(out) to convert out into the OPENCV matrix "spec_simil_img".
*/

void HyperFunctionsGPU::spec_sim_GPU() {

    parent_control<<<grid_size, block_size>>>(d_out, d_img_array, N_size, num_lay, d_ref_spectrum, spec_sim_alg);

    hipDeviceSynchronize();
    hipMemcpyAsync(out, d_out, sizeof(float) * N_size, hipMemcpyDeviceToHost); 
    hipDeviceSynchronize();

    this->oneD_array_to_mat(out);   
}

/**
 * Parent Controller for Cuda
*/
__global__ void parent_control(float *out, int *img_array, int n, int num_layers, int* ref_spectrum, int sim_alg){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    switch(sim_alg) {
    case 0:
        child_SAM(out, img_array, n, num_layers, ref_spectrum);          
        if (tid < n){
            out[tid] = 255 * out[tid];
        }
        break;
    case 1:
        child_SCM(out, img_array, n, num_layers, ref_spectrum);
        if(tid < n){
            out[tid] = (1-out[tid]) * 0.5 * 255;
        }
        break;

    case 2:

        child_SID(out, img_array, n, num_layers, ref_spectrum);
        if(tid < n){
            
            out[tid] = 60*out[tid];
        }
        break;
    case 3:
        
        child_EuD(out, img_array, n, num_layers, ref_spectrum);
        break;
    case 4:
        child_chisq(out, img_array, n, num_layers, ref_spectrum);
        if(tid < n){
            out[tid] = out[tid] * 255; 
        }
        break;
    case 5:
        child_cos(out, img_array, n, num_layers, ref_spectrum);
        break;
    case 6:
        child_cityblock(out, img_array, n, num_layers, ref_spectrum);
        break;
    case 7:
        child_JM(out, img_array, n, num_layers, ref_spectrum);
        if(tid < n){
            out[tid] = out[tid] * 255; 
        }
       
        break;
    case 8:
        //ns3
        if(tid < n){
           

           child_EuD(out, img_array, n, num_layers, ref_spectrum);
           float eud_result = out[tid];

            child_SAM(out, img_array, n, num_layers, ref_spectrum);
            float sam_result = out[tid];

           out[tid] = 6000 * sqrt(pow(sqrtf(1/num_layers) * eud_result, 2)
                                      +pow(1-cos(sam_result), 2));
            
        }
        break;
    case 9:
        //JM-SAM
        if(tid < n){

            child_JM(out, img_array, n, num_layers, ref_spectrum);
            float jm_result = out[tid];
            

            child_SAM(out, img_array, n, num_layers, ref_spectrum);
            float sam_result = out[tid];
            
            out[tid] = 255 * (jm_result * tanf(sam_result));
        }
        break;
    case 10:
        // SCA
        if(tid < n){
            child_SCM(out, img_array, n, num_layers, ref_spectrum);
            float scm_result = out[tid];

            out[tid] = 255 * ((1/3.141592654) * acos((scm_result)+1)*0.5);

        }
        break;
    case 11:
        // SID-SAM
        if (tid < n){
                
            child_SID(out, img_array, n, num_layers, ref_spectrum);
            float sid_result = out[tid];
            

            child_SAM(out, img_array, n, num_layers, ref_spectrum);
            float sam_result = out[tid];
            
            out[tid] = 255 * (sid_result * tanf(sam_result));
        }
        break;
    case 12:
        // SID-SCA
        if(tid < n){
            child_SCM(out, img_array, n, num_layers, ref_spectrum);
            float scm_result = out[tid];

            child_SID(out, img_array, n, num_layers, ref_spectrum);
            float sid_result = out[tid];

            out[tid] = 255 * sid_result * tanf(( acos((scm_result+1)*0.5)));
        }
        break;
    case 13:
        child_hellinger(out, img_array, n, num_layers, ref_spectrum);
        if(tid < n){
            out[tid] = out[tid] * 255; 
        }
        break;
    case 14:
        child_canberra(out, img_array, n, num_layers, ref_spectrum);
        if(tid < n){
            out[tid] = out[tid] * .8; 
        }
        break;
   

    default:
        printf("It Broke !!\n");
        break;

    }
}



__device__ void child_SAM(float *out, int *img_array, int n, int num_layers, int* ref_spectrum){
    
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    float sum1=0, sum2=0;
    float sum3 = 0;
    for (int a=0; a<num_layers-1; a++) {
        sum3+=ref_spectrum[a] *ref_spectrum[a]; //sum of squared reference spectra values
    }

    

    if (tid < n){
        int offset=tid*num_layers; //calculating which index in the image array the values for threadID pixel start at
        for (int a=0; a<num_layers-1; a++) //iterating through spectra layers for that pixel
        {
            sum1+=img_array[offset+a]*ref_spectrum[a]; //image spectra values * corresponding referencec spectrum values
            sum2+=img_array[offset+a]*img_array[offset+a]; //Squared image spectra values
        }
        
        if (sum1<=0 || sum2<=0 || sum3<=0 )
        {
            out[tid] = 1; // set to white due to an error
        }
        else
        {
            float temp1= sum1/(sqrt(sum2)*sqrt(sum3));
            double alpha_rad = acos(temp1);
            out[tid] = (float)alpha_rad; //(int)((double)alpha_rad*(double)255/(double)3.14159) ;
        }
    }
}



/**
 * 
 * Spectral Corellation Mapper function for spectral similarity analysis
 * 
 * 
 * 
*/
__device__ void child_SCM(float *out, int *img_array, int n, int num_layers, int* ref_spectrum){
    // parallelize tasks
    // pixels are stored with all pixel values next to eachother for the layers    
    // n is number of pixels 

    // blockID : block index within the grid
    // blockDim : how many threads per block
    // threadIdx : thread index within the block 

    int tid = blockIdx.x * blockDim.x + threadIdx.x; //unique thread ID
    float sum1=0, sum2=0, sum3=0, mean1=0, mean2=0;
    if (tid < n){
        int offset=tid*num_layers;

        for (int a=0; a<num_layers-1; a++)
        {
            mean1+=((float)1/(float)(num_layers-1)* (float)img_array[offset+a])  ;
            mean2+=((float)1/(float)(num_layers-1)* (float)ref_spectrum[a]) ;
        }

        for (int a=0; a<num_layers-1; a++)
        {
            sum1+=(img_array[offset+a]-mean1)*(ref_spectrum[a]-mean2) ;
            sum2+=(img_array[offset+a]-mean1)*(img_array[offset+a]-mean1);
            sum3+=(ref_spectrum[a]-mean2)*(ref_spectrum[a]-mean2);
        }        
        if (sum2<=0 || sum3<=0 )
        {
            out[tid] = 1; // set to white due to an error
        }
        else
        {
            float temp1= sum1/(sqrt((sum2)*(sum3)));
            // double alpha_rad=acos(temp1);
            out[tid] = temp1;
        }
    }
}


__device__ void child_cos(float *out, int *img_array, int n, int num_layers, int* ref_spectrum){

    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    float sum1=0, sum2=0;
    float sum3 = 0;
    for (int a=0; a<num_layers-1; a++) {
        sum3+=ref_spectrum[a] *ref_spectrum[a]; //sum of squared reference spectra values
    }
    if (tid < n){
        int offset=tid*num_layers; //calculating which index in the image array the values for threadID pixel start at
        for (int a=0; a<num_layers-1; a++) //iterating through spectra layers for that pixel
        {
            sum1+=img_array[offset+a]*ref_spectrum[a]; //image spectra values * corresponding referencec spectrum values
            sum2+=img_array[offset+a]*img_array[offset+a]; //Squared image spectra values
        }
        
        if (sum1<=0 || sum2<=0 || sum3<=0 )
        {
            out[tid] =255; // set to white due to an error
        }
        else
        {
            float temp1= sum1/(sqrt(sum2)*sqrt(sum3));
            double alpha_rad=acos(temp1);
            out[tid] =(int)((double)alpha_rad*(double)255) ;
        }
    }
}
/**
 * SID 
 * */

__device__ void child_SID(float *out, int *img_array, int n, int num_layers, int* ref_spectrum){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    float sum1=0, sum2=0, ref_sum=0, pix_sum=0;

    if (tid < n){
        int offset=tid*num_layers;
        for (int a=0; a<num_layers-1; a++)
        {
            if (ref_spectrum[a]<1){ref_spectrum[a]+=1;}
            if (img_array[offset+a]<1){img_array[offset+a]+=1;}              
            ref_sum+= ref_spectrum[a] ;
            pix_sum+= img_array[offset+a];
        }
        
        // error handling to avoid division by zero
        if (ref_sum<1){ref_sum+=1;}
        if (pix_sum<1){pix_sum+=1;}
        
        float ref_new[300], pix_new[300];
        
        for (int a=0; a<num_layers-1; a++)
        {
            ref_new[a]=ref_spectrum[a] / ref_sum ; //probability distribution for reference spectrum
            pix_new[a]=img_array[offset+a]/pix_sum; //probabiltiy distribution for our image
            // error handling to avoid division by zero
        }
        
        for (int a=0; a<num_layers-1; a++)
        {
            sum1+= ref_new[a]*log(ref_new[a]/pix_new[a]);
            sum2+= pix_new[a]*log(pix_new[a]/ref_new[a]);
        }        

        // need to normalize the results better here
        out[tid] =(sum1+sum2);
        // if (out[tid]>255){out[tid]=255;}

    }
}

/*JM*/
__device__ void child_JM(float *out, int *img_array, int n, int num_layers, int* ref_spectrum){

    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < n) {
        double referenceSpecIntegral = 0;
        double pixelSpecIntegral = 0;

        int offset = tid * num_layers;
        for (int i = 0; i < num_layers; i++) {
            referenceSpecIntegral += ref_spectrum[i];
            pixelSpecIntegral += img_array[offset + i];
        }

        double BC = 0;
        for (int i = 0; i < num_layers; i++) {
            BC += sqrt((ref_spectrum[i] / referenceSpecIntegral) * (img_array[offset + i] / pixelSpecIntegral));
        }

        double Bhattacharyya = -log(BC);
        double JM_distance = sqrt(2 * (1 - exp(-Bhattacharyya)));
        double JM_distance_scaled = JM_distance * 0.70711;
        out[tid] = (float)(JM_distance_scaled);
    }

}

/*EuD*/
__device__ void child_EuD(float *out, int *img_array, int n, int num_layers, int* ref_spectrum){

    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    float sum1=0, sum2=0, sum3 = 0;

    for (int a=0; a<num_layers-1; a++) {
        sum3+=ref_spectrum[a] *ref_spectrum[a]; //sum of squared reference spectra values
    }
    if (tid < n){
        int offset=tid*num_layers; //calculating which index in the image array the values for threadID pixel start at
        for (int a=0; a<num_layers-1; a++) //iterating through spectra layers for that pixel
        {
            sum1+=img_array[offset+a]*ref_spectrum[a]; //image spectra values * corresponding referencec spectrum values
            sum2+=img_array[offset+a]*img_array[offset+a]; //Squared image spectra values
        }
        
        if (sum1<=0 || sum2<=0 || sum3<=0 )
        {
            out[tid] =255; // set to white due to an error
        }
        else
        {
            float temp1= sum1/(sqrt(sum2)*sqrt(sum3));
            double alpha_rad=acos(temp1);
            out[tid] =(int)((double)alpha_rad*(double)255) ;
        }
    }
}

/**
 * City Block
*/
__device__ void child_cityblock(float *out, int *img_array, int n, int num_layers, int* ref_spectrum){

    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    float sum1=0;
    
    if (tid < n){
        int offset = tid * num_layers;
        for (int a=0; a<num_layers-1; a++) {
            sum1 += abs(img_array[offset + a] - ref_spectrum[a]);

        }
        out[tid] = sum1/(num_layers + 255);
        
        
    }

}

/**
 * Chi-squared Algorithm
*/
__device__ void child_chisq(float *out, int *img_array, int n, int num_layers, int* ref_spectrum){

    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    double referenceSpecSum = 0;
    double pixelSpecSum = 0;
    double sqrDist = 0;
    double sum = 0;
    double chiSq = 0;
    
    if (tid < n){
        int offset = tid * num_layers;
        for (int a=0; a<num_layers-1; a++) {
            referenceSpecSum += ref_spectrum[a];
            pixelSpecSum += img_array[offset + a];
        }

        for (int a=0; a<num_layers-1; a++) {
            sqrDist = pow((img_array[offset + a] - ref_spectrum[a]), 2);
            sum = (img_array[offset + a] + ref_spectrum[a]);
            chiSq += (sqrDist / sum);
        }
        
        out[tid] = chiSq; 
    }

}



/**
 * hellinger
*/
__device__ void child_hellinger(float *out, int *img_array, int n, int num_layers, int* ref_spectrum){

    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    float sum1=0, sum2=0, sumAll=0;
    double referenceSpecSum = 0;
    double pixelSpecSum = 0;
    
    if (tid < n){
        int offset = tid * num_layers;
        for (int a=0; a<num_layers-1; a++) {
            
            if (ref_spectrum[a]<1){ref_spectrum[a]+=1;}
            if (img_array[offset+a]<1){img_array[offset+a]+=1;}
            
            
            referenceSpecSum += ref_spectrum[a];
            pixelSpecSum += img_array[offset + a];
        }
        
        for (int a=0; a<num_layers-1; a++) {
            double refNew = ref_spectrum[a] / referenceSpecSum;
            double pixNew = img_array[offset+a] / pixelSpecSum;
            sum1 = sqrt(refNew); // √p_i
            sum2 = sqrt(pixNew); // √q_i
            sumAll += pow(sum1 - sum2, 2); // sum from i=1 to k (√p_i - √q_i)^2 
        }
       
      
        out[tid] = (sqrt(sumAll))/1.414; //sqrt(2)=1.41421356237
    }

}

/**
 * canberra
*/
__device__ void child_canberra(float *out, int *img_array, int n, int num_layers, int* ref_spectrum){

    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    float sum1=0;
    
    if (tid < n){
       int offset = tid * num_layers;
       for (int a=0; a<num_layers-1; a++) {
            // sum1 += abs(img_array[offset + a] - ref_spectrum[a]);
            sum1 += abs(img_array[offset + a] - ref_spectrum[a]) / (img_array[offset + a] + ref_spectrum[a]);


        }
        out[tid] = sum1;
        
        
    }

}


void HyperFunctionsGPU::deallocate_memory() 
{
    hipFree(d_ref_spectrum); hipFree(d_out); hipHostFree(out);
}

/* allocating CUDA memory. 
* In cuda, grids contain blocks of threads, which are used for parallel computations. 
* One pixel in img_array needs one thread for a computation, so threads = number of pixels. 
* Set the number of threads per block to be 512 - this is the maximum threads per block for older GPUs.
* Grid size is the number of of blocks, given by the number of threads / block size + 1. 
*/
void HyperFunctionsGPU::allocate_memory() {
    N_points=mlt1[1].rows*mlt1[1].cols*mlt1.size(); 
    N_size=mlt1[1].rows*mlt1[1].cols;    
    num_lay=  mlt1.size();
    block_size = 512;
    grid_size = ((N_points + block_size) / block_size); 
    
    int tmp_len1=reference_spectrums[ref_spec_index].size(); 

    hipHostAlloc ((void**)&out, sizeof(float) * N_size, hipHostMallocDefault);
    hipMalloc((void**)&d_out, sizeof(float) * N_size);
    hipMalloc((void**)&d_ref_spectrum, sizeof(int) * tmp_len1);

    //allocating memory on the GPU device

    ref_spectrum=new int[tmp_len1];
    for (int i=0;i<reference_spectrums[ref_spec_index].size();i++)
    {
        ref_spectrum[i] = reference_spectrums[ref_spec_index][i]; 
        //converting the 2-D array of reference spectrums into one-D for CUDA processing
    }
    hipMemcpy(d_ref_spectrum, ref_spectrum, sizeof(int) * tmp_len1, hipMemcpyHostToDevice);
    delete[] ref_spectrum;
    //copying our existing memory that holds image data and reference spectrum data to the 
    //allocated memory on the GPU
}

/**
 * Converts one-D array to 1 channel 8 bit OPENCV matrix. 
 * Used in manipulating spectral similarity data. 
*/
void HyperFunctionsGPU::oneD_array_to_mat(float* img_array)
{
    spec_simil_img = cv::Mat(mlt1[1].rows, mlt1[1].cols, CV_32FC1, img_array);
    spec_simil_img.convertTo(spec_simil_img, CV_8UC1); //converting to 8 bit unsigned, 1 channel. 
    
}

/**
 * Converting a one-D array to a OPENCV matrix. 
 * This matrix will has three channels per point which store RGB color values. 
*/

void HyperFunctionsGPU::oneD_array_to_mat(int* img_array, int cols, int rows, int channels, Mat* mlt1)
{

    *mlt1 = cv::Mat(mlt1->rows, mlt1->cols, CV_32SC3, img_array);
    mlt1->convertTo(*mlt1, CV_8UC3);
}


__global__ void mat_to_oneD_array_child(uchar* mat_array, int* img_array, int n, int start, int inc) 
{ 

    // blockID : block index within the grid
    // blockDim : how many threads per block
    // threadIdx : thread index within the block 
    int tid = blockIdx.x * blockDim.x + threadIdx.x; //unique thread ID
    if (tid < n){
       img_array[tid * inc + start] = mat_array[tid];
    }
}

void HyperFunctionsGPU::mat_to_oneD_array_parallel_parent()
{
    int array_size=mlt1[1].rows*mlt1[1].cols*mlt1.size();    
    uchar* d_mat_array; 
    int sz = mlt1[1].rows * mlt1[1].cols;
    hipHostAlloc ((void**)&d_mat_array, sizeof(uchar) * sz, hipHostMallocDefault);
    hipMalloc((void**)&d_img_array, sizeof(int) * array_size);
    int grid_size1 = (sz + block_size) / block_size;
    for (int i = 0; i < mlt1.size(); i++) {
        uchar* mat_array = (uchar*)mlt1[i].data;
        hipMemcpyAsync(d_mat_array, mat_array, sizeof(uchar) * sz, hipMemcpyHostToDevice);
        mat_to_oneD_array_child<<<grid_size1, block_size>>>(d_mat_array, d_img_array, sz, i, mlt1.size());
    }
    hipHostFree(d_mat_array);  
}

void HyperFunctionsGPU::mat_to_oneD_array_parallel_parent(vector<Mat>* matvector1)
{
    vector<Mat> matvector = *matvector1;
    uchar* d_mat_array;
    int array_size=matvector[1].rows*matvector[1].cols*matvector.size();  
    int sz = matvector[1].rows * matvector[1].cols;
    hipHostAlloc ((void**)&d_mat_array, sizeof(uchar) * sz, hipHostMallocDefault);
    hipMalloc((void**)&d_classified_img_array, sizeof(int) * array_size);
    block_size = 512;
    int grid_size1 = (sz + block_size) / block_size;
    for (int i = 0; i < matvector.size(); i++) {
        uchar* mat_array = (uchar*)matvector[i].data;
        hipMemcpy(d_mat_array, mat_array, sizeof(uchar) * sz, hipMemcpyHostToDevice);
        mat_to_oneD_array_child<<<grid_size1, block_size>>>(d_mat_array, d_classified_img_array, sz, i, matvector.size());
    }
    hipHostFree(d_mat_array);  
}

/**
 * Fills in the "out" array, which holds RGB values based on which reference spectra a pixel in the image is most similar to. 
*/

__global__ void img_test_classifier(int *out, int *img_array, int num_pixels, int num_spectrums, int* color_info, int classification_threshold ) 
{
    // blockID : block index within the grid
    // blockDim : how many threads per block
    // threadIdx : thread index within the block 

    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < num_pixels){
        //out[tid] =img_array[tid];
        int offset=tid*num_spectrums; //where the spectral similarity image starts for a pixel. 
        int low_val;
        for (int a=0; a<num_spectrums;a++) //iterating through all the spectral similarity scores for a pixel
        {
            if (a==0) 
            //Setting the initial lowest value. Goal is to look through all the spectral
            //similarity values for a pixel and find the lowest score. The reference
            //spectra that yields the lowest score will be the most similar.  
            {
                low_val=img_array[offset];
                if (low_val<=classification_threshold)
                {
                out[tid*3]=color_info[a+0];
                out[tid*3+1]=color_info[a+1];  
                out[tid*3+2]=color_info[a+2];
                }
                else
                {
                out[tid*3]=0;
                out[tid*3+1]=0;  
                out[tid*3+2]=0;
                }
            }
            else if (img_array[offset+a]<low_val && img_array[offset+a]<= classification_threshold) 
            //looking for a new minimum. If found, set the color channels of the RGB image to the color corresponding to the reference spectra. 
            {
                out[tid*3]=color_info[a*3+0];
                out[tid*3+1]=color_info[a*3+1];
                out[tid*3+2]=color_info[a*3+2];                
            }            
           
        }
    }   
}

/**
 * Takes hyperspectral data and gives each pixel a color based on which reference spectra
 * it is most similar to. 
 * 
*/

void HyperFunctionsGPU::semantic_segmentation() {
    ref_spec_index = 0;
    int tmp_len1 = reference_spectrums[0].size();
    vector<Mat> similarity_images;
    int* ref_spectrum = new int[tmp_len1];

    for (int i=0;i<reference_spectrums[ref_spec_index].size();i++)
    {
        ref_spectrum[i] = reference_spectrums[ref_spec_index][i]; 
    }
    hipMemcpy(d_ref_spectrum, ref_spectrum, sizeof(int) * tmp_len1, hipMemcpyHostToDevice);

    this->spec_sim_GPU(); 
    //performs spectral similarity, comparing each pixel in our image array to the first reference spectra. 
    similarity_images.push_back(spec_simil_img);  
    for (int i = 1; i < reference_spectrums.size(); i++) { //loop to iterate through all the reference spectras. 
        for (int j=0; j < reference_spectrums[i].size(); j++) {
            ref_spectrum[j] = reference_spectrums[i][j]; //updating the reference spectrum we are comparing our pixels to. 
        }
        //updating the memory allocated on the GPU that stores the current reference spectra
        hipMemcpy(d_ref_spectrum, ref_spectrum, sizeof(int) * tmp_len1, hipMemcpyHostToDevice); 
        this->spec_sim_GPU();
        similarity_images.push_back(spec_simil_img);
    }

    int array_size2=similarity_images[1].rows*similarity_images[1].cols*similarity_images.size();  
    //converting the vector of matrices that store the similarity values for each reference spectrum into a 1-D array

    mat_to_oneD_array_parallel_parent(&similarity_images);

    int *d_color_info, *d_out2, *out2;
    int N_size_sim = similarity_images[1].rows*similarity_images[1].cols*3; 
    int N_points_sim = similarity_images[1].rows*similarity_images[1].cols*similarity_images.size(); 
    int grid_size_sim = ((N_points + block_size) / block_size);
    int tmp_len2 = color_combos.size()*3;


    /**
     * out2 : 1-d array that represents matrix with 3 channels, to store R G and B values. 
     * d_clasified_img_array : 1-d array containing spectra similarity values 
     * d_color_info : holds reference colors that will be used to colorize our final image
     * 
    */
   
    hipHostAlloc ((void**)&out2, sizeof(int) *N_size_sim, hipHostMallocDefault); 
    hipMalloc((void**)&d_out2, sizeof(int) * N_size_sim);
    int temp_val=reference_colors.size() * 3;
    hipMalloc((void**)&d_color_info, sizeof(int) * temp_val);

    int* reference_colors_c = new int[reference_colors.size() * 3];

    //converting reference_colors into a 1-d array
    for (int i = 0; i < reference_colors.size(); i++) {
        reference_colors_c[i*3] = reference_colors[i][0];
        reference_colors_c[i*3+1] = reference_colors[i][1];
        reference_colors_c[i*3+2] = reference_colors[i][2];
    }

    hipMemcpy(d_color_info, reference_colors_c, sizeof(int) * temp_val, hipMemcpyHostToDevice);
    //multi-threaded function to find the most similar spectra for a pixel and color it based on the color assigned to that spectra
    img_test_classifier<<<grid_size_sim,block_size>>>(d_out2, d_classified_img_array, N_size_sim/3, similarity_images.size(), d_color_info,classification_threshold);

    /**
     * copying the color image into out2, and converting that into a OPENCV matrix. 
    */

    hipMemcpy(out2, d_out2, sizeof(int) * N_size_sim, hipMemcpyDeviceToHost);
   
    Mat test_img2(similarity_images[1].rows, similarity_images[1].cols, CV_8UC3, Scalar(0,0,0)); 
    oneD_array_to_mat(out2, similarity_images[1].cols,similarity_images[1].rows,3, &test_img2);
    classified_img = test_img2;

    hipFree(d_color_info);
    hipFree(d_out2);
    hipFree(d_classified_img_array);
    hipHostFree(out2);
    delete[] reference_colors_c;
    delete[] ref_spectrum;

}

